#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "gpu_add.h"


__global__
void add(int n, float *x, float *y){
    for (int i=0; i<n; i++)
    y[i] = x[i] + y[i];
}

int add_wrapper(){
    // 1M elements
    int N = 1<<20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // init x and y arrays
    for (int i=0; i<N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // run
    add<<<1, 1>>>(N, x, y);

    hipDeviceSynchronize();

    // check for error
    float maxErr = 0.0f;
    for (int i=0; i<N; i++)
        maxErr = fmax(maxErr, fabs(y[i]-3.0f));

    std::cout << "Max error from GPU: " << maxErr << std::endl;

    hipFree(x);
    hipFree(y);

    // return and exit
    return 0;
}